/* ==================================================================
	Programmer: Yicheng Tu (ytu@cse.usf.edu)
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc SDH.c -o SDH in the C4 lab machines
   ==================================================================
*/

/* USF Fall 2019 CIS4930 Programming on Massively Parallel Systems
   Project Description: Write a CUDA program to implement the same
   functionality as the CPU only code

   Student: Alexander Cook
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>


#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry{
	//float min;
	//float max;
	unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;

bucket * histogram;		/* list of all buckets in the histogram   */
bucket * z_histogram;   /* histogram initialized to all 0s        */
long long	PDH_acnt;	/* total number of data points            */
int num_buckets;		/* total number of buckets in the histogram */
double   PDH_res;		/* value of w                             */
atom * atom_list;		/* list of all data points                */

/* These are for an old way of tracking time */
struct timezone Idunno;	
struct timeval startTime, endTime;


/* 
	Distance of two points in the atom_list 
*/
double p2p_distance(int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}

//Device helper function which now takes a pointer as an argument instead of using a global pointer
__device__ double d_p2p_distance(atom *atom_list, int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


__global__ void PDH_kernel(atom *d_atom_list, bucket *d_histogram, int PDH_acnt, int PDH_res){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	int h_pos;
	double dist;

	if(i < j && i < PDH_acnt && j < PDH_acnt){ // i < j so distances are not counted twice
		dist = d_p2p_distance(d_atom_list, i,j);
			h_pos = (int) (dist / PDH_res);
			atomicAdd(&(d_histogram[h_pos].d_cnt), 1);
			 
	}
	
}

//Single threaded kernel for testing
__global__ void PDH_kernelST(atom *d_atom_list, bucket *d_histogram, int PDH_acnt, int PDH_res){
	int i = threadIdx.x;

	int j, h_pos;
	double dist;
	
	for(; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = d_p2p_distance(d_atom_list,i,j);
			h_pos = (int) (dist / PDH_res);
			d_histogram[h_pos].d_cnt++;
		} 
	}
	
}


/* 
	Brute-force SDH solution in a single CPU thread 
*/
int PDH_baseline() {
	int i, j, h_pos;
	double dist;
	
	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos].d_cnt++;
		} 
	}
	return 0;
}

/* 
	Set a checkpoint and show the (natural) running time in seconds 
*/
double report_running_time(const char* version) {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for %s version: %ld.%06ld\n",version , sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}


/* 
	Print the counts in all buckets of the histogram 
*/
void output_histogram(bucket *histogram){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

//Prints difference between two histograms
void output_histogram_diff(bucket *histo1, bucket *histo2){
	int i; 
	long long total_cnt1 = 0;
	long long total_cnt2 = 0;
	printf("Difference between CPU and GPU histogram\n");
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histo1[i].d_cnt - histo2[i].d_cnt);
		total_cnt1 += histo1[i].d_cnt;
		total_cnt2 += histo2[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt1 - total_cnt2);
		else printf("| ");
	}
}


int main(int argc, char **argv)
{
	int i;

	PDH_acnt = atoi(argv[1]);
	PDH_res	 = atof(argv[2]);

	//Allocate host memory
	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);
	z_histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);
	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	//initialize histogram to zero
	memset(z_histogram, 0, sizeof(bucket)*num_buckets);

	//Allocate device memory
	bucket *d_histogram; //pointer to array of buckets
	atom *d_atom_list; //pointer to array of atoms

	hipMalloc((void**)&d_histogram, sizeof(bucket)*num_buckets);
	hipMalloc((void**)&d_atom_list, sizeof(atom)*PDH_acnt);

	
	srand(1);
	/* Generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}

	//Copy host data to device memory
	hipMemcpy(d_histogram, z_histogram, sizeof(bucket)*num_buckets, hipMemcpyHostToDevice);
	hipMemcpy(d_atom_list, atom_list, sizeof(atom)*PDH_acnt, hipMemcpyHostToDevice);
	
	/* Start counting time */
	gettimeofday(&startTime, &Idunno);
	
	/* Call CPU single thread version to compute the histogram */
	PDH_baseline();
	
	/* Report running time for CPU version */ 
	report_running_time("CPU");
	
	/* Print out the histogram */
	output_histogram(histogram);


	//Define 2D block and grid size
	int num_threads = 16; //number of threads in one dimension of a block
	dim3 blockDim(num_threads,num_threads); //num_threads^2 threads per block
	int num_blocks = (PDH_acnt + num_threads - 1)/num_threads; //calculate number of blocks for the grid in a particular dimension
	dim3 gridDim(num_blocks, num_blocks); //the grid is the same size in x and y dimension
	
	//Start counting time
	gettimeofday(&startTime, &Idunno);

	//Launch kernel
	PDH_kernel<<<gridDim,blockDim>>>(d_atom_list, d_histogram, PDH_acnt, PDH_res);
	//PDH_kernelST<<<1,1>>>(d_atom_list, d_histogram, PDH_acnt, PDH_res);

	//Copy data from gpu memory to host memory
	bucket * GPU_histogram;
	GPU_histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);
	hipMemcpy(GPU_histogram, d_histogram, sizeof(bucket)*num_buckets, hipMemcpyDeviceToHost);
	
	//Report GPU running time
	report_running_time("GPU");

	/* Print out the histogram again for gpu version */
	output_histogram(GPU_histogram);

	//print difference between the two histograms
	output_histogram_diff(histogram, GPU_histogram);

	free(histogram);
	free(atom_list);
	free(GPU_histogram);
	hipFree(d_histogram);
	hipFree(d_atom_list);
	

	return 0;
}


